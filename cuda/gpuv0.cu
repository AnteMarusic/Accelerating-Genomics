#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h> // Include the string.h header for strcspn function
#include <limits.h>

#define TRUE 1
#define FALSE 0

//#undef DEBUG
#define DEBUG

#ifdef DEBUG
#define PRINT printf
#else
#define PRINT // macros
#endif

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

#define NEGATIVE_INFINITY INT_MIN  // Minimum value for an int
#define MAX_SIZE 10000
#define SCORE_MATCH 1
#define SCORE_MISMATCH (-1)
#define SCORE_OPEN_GAP (-3)
#define SCORE_EXTEND_GAP (-1)
#define P 1
#define Q 2
#define D 3


/*
Example of how the strings are in the matrixes
s1 = CG
s2 = CCGA
nx is the size s2 + 1
    _CCGA nx
_   
C    0000
G    0000
ny
*/

__device__ void a_print(int *a, int len, int nx) {
    int k = 1;
    int q = 1;
    printf("\n____________________________\n");
    for(int i = 0; i < len; i ++) {
        a[i] == NEGATIVE_INFINITY ? printf("-∞,") : printf("%d,", a[i]);
        if (i + 1 == (nx) * k) {
            if (i + 1 == (nx * 3) * q) {
                printf("\n");
                q++;
            }
            printf("|");
            k ++;
        }
    }
    printf("\n____________________________\n");
    return;
}


__device__ int max_of_two_integers(int a, int b) {
    return (a > b) ? a : b;
}

__device__ int sum_with_infinity (int a, int b) {
    return (a == NEGATIVE_INFINITY || b == NEGATIVE_INFINITY) ? NEGATIVE_INFINITY : a + b;
}

__device__ int max_of_four_integers (int a, int b, int c, int d) {
    return max_of_two_integers (max_of_two_integers (a, b), max_of_two_integers(c, d));
}

void reverse_array(char arr[], int size) {
  // Loop through half the array
  for (int i = 0; i < size / 2; i++) {
    // Swap elements at opposite positions
    char temp = arr[i];
    arr[i] = arr[size - 1 - i];
    arr[size - 1 - i] = temp;
  }
}

//returns positive values in case of success and the value is in val
//returns negative number in case of failure
__device__ int m_get(int *val, int iy, int ix, int mat, int *antidiags, int nx, int ny) {
    int antid_dim = nx;
    int offset_antid = 0;
    int offset_mat = 0;
    int index = 0;
    if (ix >= nx || iy >= ny|| ix < 0 || iy < 0) {
        printf("[m_get thread %d] out of bounds set iy: %d, ix: %d\n",threadIdx.x, iy, ix);
        return -1;
    }
    /*
    if (ix + iy != cur_antid_num) {
        printf("[m_get] iy: %d, ix: %d not in mem\n", iy, ix);
        return -1;
    }
    */
    
    switch (mat) {
        case P :
            offset_mat = 0;
            break;
        case Q :
            offset_mat = antid_dim*3;
            break;
        case D :
            offset_mat = antid_dim*2*3;
            break;
        default :
            printf("[m_get thread %d] mat doesn't exist\n", threadIdx.x);
    }
    int cur_antid_num = ix + iy;
    offset_antid = cur_antid_num % 3;//three is the num of antid saved at the same time
    
    if(cur_antid_num > ny-1){
        index = ny - 1 - iy + offset_antid*antid_dim + offset_mat;
    }
    else {
        index = ix + offset_antid*antid_dim + offset_mat;
    }
    *val = antidiags[index];
    printf("[m_get thread %d] getting val: %d, at iy:%d, ix:%d, on matrix: %d, at index: %d\n", threadIdx.x, *val, iy, ix, mat, index);
    return 1;
}

//returns positive values in case of success and the value is in val
//returns negative number in case of failure
__device__ int m_set(int *val, int iy, int ix, int mat, int *antidiags, int nx, int ny) {
    int antid_dim = nx;
    int offset_antid = 0;
    int offset_mat = 0;
    int index = 0;

    //check the input
    if (ix >= nx || iy >= ny|| ix < 0 || iy < 0) {
        printf("[m_set thread %d] out of bounds set iy: %d, ix: %d\n", threadIdx.x, iy, ix);
        return -1;
    }
    /*if (ix + iy != cur_antid_num) {
        printf("[m_set] iy:%d, ix:%d not in mem\n", iy, ix);
        return -1;
    }*/
    
    //compute the offset_mat
    switch (mat) {
        case P :
            offset_mat = 0;
            break;
        case Q :
            offset_mat = antid_dim*3;
            break;
        case D :
            offset_mat = antid_dim*2*3;
            break;
        default :
            printf("[m_set thread %d] mat doesn't exist\n", threadIdx.x);
    }
    int cur_antid_num = ix + iy;
    //three is the num of antid saved at the same time
    offset_antid = cur_antid_num % 3;
    if(cur_antid_num > ny-1){
        index = ny - 1 - iy + offset_antid*antid_dim + offset_mat;
    }
    else {
        index = ix + offset_antid*antid_dim + offset_mat;
    }

    printf("[m_set thread %d] setting val: %d, iy:%d, ix:%d, at index: %d on matrix %d\n", threadIdx.x, *val, iy, ix, index, mat);
    antidiags[index] = *val;
    return 1;
}


__global__ void alignGPU(char *d_sx, char *d_sy, int *d_result, int *d_error, int antidiags_size, int sx_len, int sy_len, int nx, int ny, int match_score, int mismatch_score, int gap_open_score, int gap_extend_score) {
    extern __shared__ int s[];
    int *antidiags = s;
    int *shared_max = &s[antidiags_size];
    //__shared__ int max; //check if I can do this. How to collect the max value between all threads? I can do it at the end I think
    int num_of_threads = blockDim.x*blockDim.y*blockDim.z;
    //printf("num_of_threads: %d\n", num_of_threads);
    int antid_num = nx + ny - 1;
    int yr_phase = nx - 1;//yellow is the phase where the cur_antid_dim less then the max possible and is growing
    //with each iteration. red is the same but cur_antid_dim is decreasing. red and yellow have the same size.
    int o_phase = ny - nx + 1;//orange phase is the one where the cur_antid_dim is the max possible
    int err;
    int val = 1;
    int minus_infty = NEGATIVE_INFINITY;
    int zero=0;
    int iy = 0, ix = 0;
    int cur_antid_dim;
    int gen_iy = 0;
    int gen_ix = 0;
    int temp1,temp2,temp3;
    //if (threadIdx.x == 0) max = 0;
    shared_max[threadIdx.x] = 0;
    for(int cur_antid_num = 0; cur_antid_num < antid_num; cur_antid_num  ++) {
        //compute dimension of the current antidiagonal
        if(cur_antid_num <= yr_phase) { //at index yr_phase starts the o_phase
            //printf("[kernel %d] growing phase\n", threadIdx.x);
            cur_antid_dim = cur_antid_num + 1;
        }
        else if(cur_antid_num >= yr_phase + o_phase) {
            //printf("[kernel %d] decreasing phase\n", threadIdx.x);
            cur_antid_dim = cur_antid_dim - 1;
        }
        else {
            //printf("[kernel] const phase\n");
        }
        //printf("[kernel %d] cur_antid_num: %d, cur_antid_dim: %d\n", threadIdx.x, cur_antid_num, cur_antid_dim);
        //compute the start of ix for the current antidiagonal
        if(cur_antid_num >= yr_phase + o_phase) {
            gen_ix ++;
        }
        iy = gen_iy - threadIdx.x;
        ix = gen_ix + threadIdx.x;
        while(true) {
            /*
            if (ix + gen_ix >= cur_antid_dim || iy - gen_iy < 0) {
                printf("out of bounds iy = %d, ix = %d\n", iy, ix);
                break;
            }
            */
            if (ix < 0 || iy < 0 || ix >= nx || iy >= ny) {
                printf("out of bounds iy = %d, ix = %d\n", iy, ix);
                break;
            }
            
            //set the first row
            if (iy == 0) {
                err = m_set(&minus_infty, iy, ix, P, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_set(&zero, iy, ix, Q, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_set(&zero, iy, ix, D, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
            }
            //set the first col
            else if(ix==0){
                err = m_set(&zero, iy, ix, P, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_set(&minus_infty, iy, ix, Q, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_set(&zero, iy, ix, D, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
            }
            //dynamic programming
            else {
                err = m_get(&temp1, iy-1, ix, D, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_get(&temp2, iy-1, ix, P, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                int b = max_of_two_integers(sum_with_infinity(temp1, SCORE_OPEN_GAP + SCORE_EXTEND_GAP), sum_with_infinity(temp2, SCORE_EXTEND_GAP));
                err = m_set(&b, iy, ix, P, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}

                err = m_get(&temp1, iy, ix-1, D, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_get(&temp2, iy, ix-1, Q, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                int a = max_of_two_integers(sum_with_infinity(temp1, SCORE_OPEN_GAP + SCORE_EXTEND_GAP), sum_with_infinity(temp2, SCORE_EXTEND_GAP));
                err = m_set(&a, iy, ix, Q, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}

                err = m_get(&temp1, iy, ix, P, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_get(&temp2, iy, ix, Q, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                err = m_get(&temp3, iy-1, ix-1, D, antidiags, nx, ny);
                if(err<1) {*d_error = -1; return;}
                //temp3 = d_sy[iy - 1] == d_sx[ix - 1] ? temp3 + SCORE_MATCH : temp3 + SCORE_MISMATCH;
                //memorise the string in opposite direction to increase locality
                temp3 = d_sy[sy_len - iy] == d_sx[ix - 1] ? temp3 + SCORE_MATCH : temp3 + SCORE_MISMATCH;
                val = max_of_four_integers(temp1, temp2, temp3, 0);
                err = m_set(&val, iy, ix, D, antidiags, nx, ny);
                shared_max[threadIdx.x] = val > shared_max[threadIdx.x] ? val : shared_max[threadIdx.x];
            }
            //update the matrix indexes
            iy -= num_of_threads;
            ix += num_of_threads;
            __syncthreads();

        }
        //move the starting point in case we are in the growing part of the matrix
        if (gen_iy < ny - 1) {
            gen_iy ++;
        }
        if(threadIdx.x == 0) {
            printf("cur_antd_num: %d\n", cur_antid_num);
            a_print(antidiags, antidiags_size, nx);
        }
    }

    //COMPUTE MAX
    for (int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s && threadIdx.x + s < blockDim.x * blockDim.y) {
        shared_max[threadIdx.x] = fmaxf(shared_max[threadIdx.x], shared_max[threadIdx.x + s]);
    }
    __syncthreads();
    }
    // The first thread in the block writes the result to global memory
    if (threadIdx.x == 0) {
        atomicMax(d_result, shared_max[0]);
    }
    //if(threadIdx.x == 0) printf("max: %d\n", max);
    //*d_result = max;
}


//todo maybe are necessary controls on the length of the input
//todo what happens if _ are present in the input string?
//int main()
int main(int argc, char *argv[]) {
     // set up device
    int dev = 1;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[main] Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    if (argc != 3) {
        printf("Usage: %s <string1> <string2>\n", argv[0]);
        return 1;
    }
    char *h_sx = argv[1];
    char *h_sy = argv[2];
    int sx_len = strlen(h_sx);
    int sy_len = strlen(h_sy);

    if (sx_len > sy_len) {
        char *tmp_sx = (char *)malloc(sx_len+1*sizeof(int));
        char *tmp_sy = (char *)malloc(sy_len+1*sizeof(int));
        strncpy(tmp_sx, h_sx, sx_len);
        strncpy(tmp_sy, h_sy, sy_len);
        h_sy = (char *)malloc(sx_len+1*sizeof(int));
        h_sx = (char *)malloc(sy_len+1*sizeof(int));
        strncpy(h_sx, tmp_sy, sy_len);
        strncpy(h_sy, tmp_sx, sx_len);
        int tmp = sx_len;
        sx_len = sy_len;
        sy_len = tmp;
        free(tmp_sx);
        free(tmp_sy);
    }
    
    printf("[main] sx: %s, xy: %s, of size %d, %d respectively\n", h_sx, h_sy, sx_len, sy_len);
    int nx = sx_len+1;
    int ny = sy_len+1;
    printf("[main] nx: %d, ny: %d\n", nx, ny);
    reverse_array(h_sy, sy_len);
    printf("[main] sx: %s, xy: %s, of size %d, %d respectively\n", h_sx, h_sy, sx_len, sy_len);

    double iStart;
    double iElaps;
    char *d_sx;
    char *d_sy;
    int *d_result;
    int *d_error;
    CHECK(hipMalloc((void **)&d_sx, sx_len));
    CHECK(hipMalloc((void **)&d_sy, sy_len));
    CHECK(hipMalloc((void **)&d_result, sizeof(int)));
    CHECK(hipMalloc((void **)&d_error, sizeof(int)));
    CHECK(hipMemcpy(d_sx, h_sx, sx_len, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_sy, h_sy, sy_len, hipMemcpyHostToDevice));
    int antidiags_size = nx * 3 * 3;
    int nBytes = antidiags_size * sizeof(int);

    dim3 block(32);
    dim3 grid(1);
    //printf("[main] GPU invocation\n");
    iStart = seconds();
    int num_of_threads = block.x*block.y*block.z;
    alignGPU <<<grid, block, nBytes+ num_of_threads*sizeof(int)>>>(d_sx, d_sy, d_result, d_error, antidiags_size, sx_len, sy_len, nx, ny, SCORE_MATCH, SCORE_MISMATCH, SCORE_OPEN_GAP, SCORE_EXTEND_GAP); 
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("[main] alignGPU <<<(%d,%d), (%d,%d), %d>>> elapsed %f sec\n", grid.x, grid.y, block.x, block.y, nBytes, iElaps);
    printf("elapsed %f\n", iElaps);
    // check kernel error
    CHECK(hipGetLastError());

    //Copy data back from GPU memory to CPU memory.
    int *h_result = (int *)malloc(sizeof(int));
    CHECK(hipMemcpy(h_result, d_result, sizeof(int), hipMemcpyDeviceToHost));
    printf("Score: %d\n", *h_result);
    int *h_error = (int *)malloc(sizeof(int));
    CHECK(hipMemcpy(h_error, d_error, sizeof(int), hipMemcpyDeviceToHost));
    //printf("[main] Error: %d\n", *h_error);

    // free device global memory
    CHECK(hipFree(d_result));
    CHECK(hipFree(d_sy));
    CHECK(hipFree(d_sx));

    // reset device
    CHECK(hipDeviceReset());
    return 0;
}
